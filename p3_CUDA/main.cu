#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <string.h>
#include "complex.h"
#include "input_image.h"
#include <thread>

using std::vector;

struct blockData
{
    int width;
    int height;
    int expOp;
    Complex sumOp;
};

struct ComplexCUDA
{
    float real;
    float imag;
};

__global__ void blockDftHoriz(struct ComplexCUDA *dftData, struct ComplexCUDA *indata, struct blockData *hbd)
{
    __shared__ struct blockData bd;
    bd = *hbd;
    __shared__ Complex data[bd.width];
    __shared__ double angle[bd.width];
    __shared__ Complex expTerm[bd.width];
    __shared__ Complex sum[bd.width];

    data[threadIdx.x] = indata[blockIdx.x*bd.width+threadIdx.x];
    data[threadIdx.x+(bd.width+1)/2] = indata[blockIdx.x*bd.width+threadIdx.x+(bd.width+1)/2];
        
    for (int t = 0; t < bd.width; t++)
    {
        angle[threadIdx.x] = bd.expOp * 2.0 * M_PI * float(t) * float(threadIdx.x) / float(bd.width);
        expTerm[threadIdx.x] = Complex(cos(angle[threadIdx.x]), sin(angle[threadIdx.x]));
        sum[threadIdx.x] = sum[threadIdx.x] + data[t] * expTerm[threadIdx.x];

        if (threadIdx.x+(bd.width+1)/2 < bd.width) 
        {
            angle[threadIdx.x+(bd.width+1)/2] = td->expOp * 2.0 * M_PI * float(t) * float(threadIdx.x+(bd.width+1)/2) / float(bd.width);
            expTerm[threadIdx.x+(bd.width+1)/2] = Complex(cos(angle[threadIdx.x+(bd.width+1)/2]), sin(angle[threadIdx.x+(bd.width+1)/2]));
            sum[threadIdx.x+(bd.width+1)/2] = sum[threadIdx.x+(bd.width+1)/2] + data[t+(bd.width+1)/2] * bd.expTerm[threadIdx.x+(bd.width+1)/2];
        }
    }
    dftData[blockIdx.x * bd.width + threadIdx.x] = sum[threadIdx.x] * bd.sumOp;
    if (threadIdx.x+(bd.width+1)/2 < bd.width) 
    {
        dftData[blockIdx.x * bd.width + threadIdx.x+(bd.width+1)/2] = sum[threadIdx.x+(bd.width+1)/2] * bd.sumOp;
    }
}


__global__ void blockDftVert(Complex *dftData, Complex *indata, struct blockData *hbd)
{
    __shared__ struct blockData bd;
    bd = *hbd;
    __shared__ Complex data[bd.height];
    __shared__ double angle[bd.height];
    __shared__ Complex expTerm[bd.height];
    __shared__ Complex sum[bd.height];

    data[threadIdx.x] = indata[threadIdx.x*bd.width+blockIdx.x];
    data[threadIdx.x+(bd.height+1)/2] = indata[(threadIdx.x+(bd.height+1)/2)*bd.width+blockIdx.x];

    for (int t = 0; t < bd.height; t++)
    {
        angle[threadIdx.x] = bd.expOp * 2.0 * M_PI * float(t) * float(threadIdx.x) / float(bd.height);
        expTerm[threadIdx.x] = Complex(cos(angle[threadIdx.x]), sin(angle[threadIdx.x]));
        sum[threadIdx.x] = sum[threadIdx.x] + data[t] * expTerm[threadIdx.x];

        if (threadIdx.x+(bd.height+1)/2 < bd.height) 
        {
            angle[threadIdx.x+(bd.height+1)/2] = td->expOp * 2.0 * M_PI * float(t) * float(threadIdx.x+(bd.height+1)/2) / float(bd.height);
            expTerm[threadIdx.x+(bd.height+1)/2] = Complex(cos(angle[threadIdx.x+(bd.height+1)/2]), sin(angle[threadIdx.x+(bd.height+1)/2]));
            sum[threadIdx.x+(bd.height+1)/2] = sum[threadIdx.x+(bd.height+1)/2] + data[t+(bd.height+1)/2] * bd.expTerm[threadIdx.x+(bd.height+1)/2];
        }
    }
    dftData2[threadIdx.x * bd.width + blockIdx.x] = sum[threadIdx.x] * bd.sumOp;
    if (threadIdx.x+(bd.height+1)/2 < bd.height) 
    {
        dftData2[(threadIdx.x+(bd.height+1)/2) * bd.width + blockIdx.x] = sum[threadIdx.x+(bd.height+1)/2] * bd.sumOp;
    }
}

/**
 * Do 2d dft in one thread. If forward is false, the inverse will be done
 * (forward is the default, though)
 */
Complex *doDft(Complex *data, int width, int height, bool forward = true)
{
    int expOp = forward ? -1 : 1;
    Complex sumOp = forward ? Complex(1.0) : Complex(float(1.0 / width));
    Complex *dftData2 = new Complex[width * height];

    struct blockData bd;

    Complex *d_data;
    Complex *d_dftData;
    Complex *d_dftData2;
    struct blockData *d_bd;

    bd.width = width;
    bd.height = height;
    bd.expOp = expOp;
    bd.sumOp = sumOp;

    hipMalloc((void **) &d_data, sizeof(Complex[width*height]));
    hipMalloc((void **) &d_dftData, sizeof(Complex[width*height]));
    hipMalloc((void **) &d_dftData2, sizeof(Complex[width*height]));
    hipMalloc((void **) &d_bd, sizeof(bd));

    hipMemcpy(d_data, data, sizeof(Complex[width*height]), hipMemcpyHostToDevice);
    hipMemcpy(d_bd, bd, sizeof(bd), hipMemcpyHostToDevice);
    
    blockDtfHoriz<<<height,(width+1)/2>>>(d_dftData,d_data,d_bd);

    blockDftVert<<<width,(height+1)/2>>>(d_dftData2,d_dftData,d_bd);

    hipMemcpy(dftData2, d_dftData2, sizeof(Complex[width*height]), hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_dftData);
    hipFree(d_dftData2);
    hipFree(d_bd);

    return dftData2;
}

int main(int argc, char **argv)
{
    if (argc != 4)
    {
        std::cout << "wrong # inputs" << std::endl;
        return -1;
    }
    bool isForward(!strcmp(argv[1], "forward"));
    char *inputFile = argv[2];
    char *outputFile = argv[3];

    if (isForward)
    {
        std::cout << "doing forward" << std::endl;
    }
    else
    {
        std::cout << "doing reverse" << std::endl;
    }
    std::cout << inputFile << std::endl;
    InputImage im(inputFile);
    int width = im.get_width();
    int height = im.get_height();

    Complex *data = im.get_image_data();
    Complex *dftData = doDft(data, width, height, isForward);

    std::cout << "writing" << std::endl;
    im.save_image_data(outputFile, dftData, width, height);
    std::cout << "dunzo" << std::endl;
    return 0;
}
